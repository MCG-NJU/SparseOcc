#include "hip/hip_runtime.h"
/*!
* Modified from Deformable DETR
*/

#include <cstdio>
#include <algorithm>
#include <cstring>
#include <hip/hip_runtime.h>
#include <>
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCAtomics.cuh>

#define CUDA_KERNEL_LOOP(i, n)                          \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;   \
      i < (n);                                          \
      i += blockDim.x * gridDim.x)

#define CUDA_NUM_THREADS 512
#define MAX_POINT 32

inline int GET_BLOCKS(const int N, const int num_threads) {
    return (N + num_threads - 1) / num_threads;
}

__device__ float ms_deform_attn_im2col_bilinear(
    const float*& bottom_data,
    const int& height, const int& width, const int& channels,
    const float& h, const float& w, const int& c) {

    const int h_low = floor(h);
    const int w_low = floor(w);
    const int h_high = h_low + 1;
    const int w_high = w_low + 1;

    const float lh = h - h_low;
    const float lw = w - w_low;
    const float hh = 1 - lh, hw = 1 - lw;

    const int w_stride = channels;
    const int h_stride = width * w_stride;
    const int h_low_ptr_offset = h_low * h_stride;
    const int h_high_ptr_offset = h_low_ptr_offset + h_stride;
    const int w_low_ptr_offset = w_low * w_stride;
    const int w_high_ptr_offset = w_low_ptr_offset + w_stride;

    float v1 = 0;
    if (h_low >= 0 && w_low >= 0) {
        const int ptr1 = h_low_ptr_offset + w_low_ptr_offset + c;
        v1 = bottom_data[ptr1];
    }
    float v2 = 0;
    if (h_low >= 0 && w_high <= width - 1) {
        const int ptr2 = h_low_ptr_offset + w_high_ptr_offset + c;
        v2 = bottom_data[ptr2];
    }
    float v3 = 0;
    if (h_high <= height - 1 && w_low >= 0) {
        const int ptr3 = h_high_ptr_offset + w_low_ptr_offset + c;
        v3 = bottom_data[ptr3];
    }
    float v4 = 0;
    if (h_high <= height - 1 && w_high <= width - 1) {
        const int ptr4 = h_high_ptr_offset + w_high_ptr_offset + c;
        v4 = bottom_data[ptr4];
    }

    const float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
    const float val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

    return val;
}

__global__ void ms_deformable_im2col_gpu_kernel_c2345(
    const float* feat_c2,
    const float* feat_c3,
    const float* feat_c4,
    const float* feat_c5,
    const int h_c2, const int w_c2,
    const int h_c3, const int w_c3,
    const int h_c4, const int w_c4,
    const int h_c5, const int w_c5,
    const float* data_sampling_loc,
    const float* data_attn_weight,
    const int batch_size,
    const int channels,
    const int num_views,
    const int num_query,
    const int num_point,
    float* data_col) {

    float res[MAX_POINT];

    CUDA_KERNEL_LOOP(index, batch_size * num_query * channels) {  // n: bs x query x channels
        int _temp = index;
        const int c_col = _temp % channels;
        _temp /= channels;
        const int sampling_index = _temp;
        _temp /= num_query;
        const int b_col = _temp;

        for (int p_col = 0; p_col < num_point; ++p_col) { res[p_col] = 0; }

        for (int p_col = 0; p_col < num_point; ++p_col) {
            // Sampling location in range [0, 1]
            int data_loc_ptr = sampling_index * num_point * 3 + p_col * 3;
            const float loc_w = data_sampling_loc[data_loc_ptr];
            const float loc_h = data_sampling_loc[data_loc_ptr + 1];
            const int loc_v = round(data_sampling_loc[data_loc_ptr + 2] * (num_views - 1));

            // Attn weights
            int data_weight_ptr = sampling_index * num_point * 4 + p_col * 4;
            const float weight_c2 = data_attn_weight[data_weight_ptr];
            const float weight_c3 = data_attn_weight[data_weight_ptr + 1];
            const float weight_c4 = data_attn_weight[data_weight_ptr + 2];
            const float weight_c5 = data_attn_weight[data_weight_ptr + 3];

            //const float h_im = loc_h * spatial_h - 0.5;  // align_corners = False
            //const float w_im = loc_w * spatial_w - 0.5;

            // C2 Feature
            float h_im = loc_h * (h_c2 - 1);  // align_corners = True
            float w_im = loc_w * (w_c2 - 1);

            if (h_im > -1 && w_im > -1 && h_im < h_c2 && w_im < w_c2) {
                const float* feat_c2_ptr = feat_c2 + b_col * num_views * h_c2 * w_c2 * channels + loc_v * h_c2 * w_c2 * channels;
                res[p_col] += ms_deform_attn_im2col_bilinear(feat_c2_ptr, h_c2, w_c2, channels, h_im, w_im, c_col) * weight_c2;
            }

            // C3 Feature
            h_im = loc_h * (h_c3 - 1);  // align_corners = True
            w_im = loc_w * (w_c3 - 1);

            if (h_im > -1 && w_im > -1 && h_im < h_c3 && w_im < w_c3) {
                const float* feat_c3_ptr = feat_c3 + b_col * num_views * h_c3 * w_c3 * channels + loc_v * h_c3 * w_c3 * channels;
                res[p_col] += ms_deform_attn_im2col_bilinear(feat_c3_ptr, h_c3, w_c3, channels, h_im, w_im, c_col) * weight_c3;
            }

            // C4 Feature
            h_im = loc_h * (h_c4 - 1);  // align_corners = True
            w_im = loc_w * (w_c4 - 1);

            if (h_im > -1 && w_im > -1 && h_im < h_c4 && w_im < w_c4) {
                const float* feat_c4_ptr = feat_c4 + b_col * num_views * h_c4 * w_c4 * channels + loc_v * h_c4 * w_c4 * channels;
                res[p_col] += ms_deform_attn_im2col_bilinear(feat_c4_ptr, h_c4, w_c4, channels, h_im, w_im, c_col) * weight_c4;
            }

            // C5 Feature
            h_im = loc_h * (h_c5 - 1);  // align_corners = True
            w_im = loc_w * (w_c5 - 1);

            if (h_im > -1 && w_im > -1 && h_im < h_c5 && w_im < w_c5) {
                const float* feat_c5_ptr = feat_c5 + b_col * num_views * h_c5 * w_c5 * channels + loc_v * h_c5 * w_c5 * channels;
                res[p_col] += ms_deform_attn_im2col_bilinear(feat_c5_ptr, h_c5, w_c5, channels, h_im, w_im, c_col) * weight_c5;
            }
        }

        for (int p_col = 0; p_col < num_point; ++p_col) {
            float* data_col_ptr = data_col + index * num_point + p_col;
            *data_col_ptr = res[p_col];
        }
    }
}

__global__ void ms_deformable_im2col_gpu_kernel_c23456(
    const float* feat_c2,
    const float* feat_c3,
    const float* feat_c4,
    const float* feat_c5,
    const float* feat_c6,
    const int h_c2, const int w_c2,
    const int h_c3, const int w_c3,
    const int h_c4, const int w_c4,
    const int h_c5, const int w_c5,
    const int h_c6, const int w_c6,
    const float* data_sampling_loc,
    const float* data_attn_weight,
    const int batch_size,
    const int channels,
    const int num_views,
    const int num_query,
    const int num_point,
    float* data_col) {

    float res[MAX_POINT];

    CUDA_KERNEL_LOOP(index, batch_size * num_query * channels) {  // n: bs x query x channels
        int _temp = index;
        const int c_col = _temp % channels;
        _temp /= channels;
        const int sampling_index = _temp;
        _temp /= num_query;
        const int b_col = _temp;

        for (int p_col = 0; p_col < num_point; ++p_col) { res[p_col] = 0; }

        for (int p_col = 0; p_col < num_point; ++p_col) {
            // Sampling location in range [0, 1]
            int data_loc_ptr = sampling_index * num_point * 3 + p_col * 3;
            const float loc_w = data_sampling_loc[data_loc_ptr];
            const float loc_h = data_sampling_loc[data_loc_ptr + 1];
            const int loc_v = round(data_sampling_loc[data_loc_ptr + 2] * (num_views - 1));

            // Attn weights
            int data_weight_ptr = sampling_index * num_point * 5 + p_col * 5;
            const float weight_c2 = data_attn_weight[data_weight_ptr];
            const float weight_c3 = data_attn_weight[data_weight_ptr + 1];
            const float weight_c4 = data_attn_weight[data_weight_ptr + 2];
            const float weight_c5 = data_attn_weight[data_weight_ptr + 3];
            const float weight_c6 = data_attn_weight[data_weight_ptr + 4];

            //const float h_im = loc_h * spatial_h - 0.5;  // align_corners = False
            //const float w_im = loc_w * spatial_w - 0.5;

            // C2 Feature
            float h_im = loc_h * (h_c2 - 1);  // align_corners = True
            float w_im = loc_w * (w_c2 - 1);

            if (h_im > -1 && w_im > -1 && h_im < h_c2 && w_im < w_c2) {
                const float* feat_c2_ptr = feat_c2 + b_col * num_views * h_c2 * w_c2 * channels + loc_v * h_c2 * w_c2 * channels;
                res[p_col] += ms_deform_attn_im2col_bilinear(feat_c2_ptr, h_c2, w_c2, channels, h_im, w_im, c_col) * weight_c2;
            }

            // C3 Feature
            h_im = loc_h * (h_c3 - 1);  // align_corners = True
            w_im = loc_w * (w_c3 - 1);

            if (h_im > -1 && w_im > -1 && h_im < h_c3 && w_im < w_c3) {
                const float* feat_c3_ptr = feat_c3 + b_col * num_views * h_c3 * w_c3 * channels + loc_v * h_c3 * w_c3 * channels;
                res[p_col] += ms_deform_attn_im2col_bilinear(feat_c3_ptr, h_c3, w_c3, channels, h_im, w_im, c_col) * weight_c3;
            }

            // C4 Feature
            h_im = loc_h * (h_c4 - 1);  // align_corners = True
            w_im = loc_w * (w_c4 - 1);

            if (h_im > -1 && w_im > -1 && h_im < h_c4 && w_im < w_c4) {
                const float* feat_c4_ptr = feat_c4 + b_col * num_views * h_c4 * w_c4 * channels + loc_v * h_c4 * w_c4 * channels;
                res[p_col] += ms_deform_attn_im2col_bilinear(feat_c4_ptr, h_c4, w_c4, channels, h_im, w_im, c_col) * weight_c4;
            }

            // C5 Feature
            h_im = loc_h * (h_c5 - 1);  // align_corners = True
            w_im = loc_w * (w_c5 - 1);

            if (h_im > -1 && w_im > -1 && h_im < h_c5 && w_im < w_c5) {
                const float* feat_c5_ptr = feat_c5 + b_col * num_views * h_c5 * w_c5 * channels + loc_v * h_c5 * w_c5 * channels;
                res[p_col] += ms_deform_attn_im2col_bilinear(feat_c5_ptr, h_c5, w_c5, channels, h_im, w_im, c_col) * weight_c5;
            }

            // C6 Feature
            h_im = loc_h * (h_c6 - 1);  // align_corners = True
            w_im = loc_w * (w_c6 - 1);

            if (h_im > -1 && w_im > -1 && h_im < h_c6 && w_im < w_c6) {
                const float* feat_c6_ptr = feat_c6 + b_col * num_views * h_c6 * w_c6 * channels + loc_v * h_c6 * w_c6 * channels;
                res[p_col] += ms_deform_attn_im2col_bilinear(feat_c6_ptr, h_c6, w_c6, channels, h_im, w_im, c_col) * weight_c6;
            }
        }

        for (int p_col = 0; p_col < num_point; ++p_col) {
            float* data_col_ptr = data_col + index * num_point + p_col;
            *data_col_ptr = res[p_col];
        }
    }
}

void ms_deformable_im2col_cuda_c2345(
    const float* feat_c2,
    const float* feat_c3,
    const float* feat_c4,
    const float* feat_c5,
    const int h_c2, const int w_c2,
    const int h_c3, const int w_c3,
    const int h_c4, const int w_c4,
    const int h_c5, const int w_c5,
    const float* data_sampling_loc,
    const float* data_attn_weight,
    const int batch_size,
    const int channels,
    const int num_views,
    const int num_query,
    const int num_point,
    float* data_col) {

    const int num_kernels = batch_size * num_query * channels;
    const int num_threads = CUDA_NUM_THREADS;

    ms_deformable_im2col_gpu_kernel_c2345 <<<GET_BLOCKS(num_kernels, num_threads), num_threads>>> (
        feat_c2, feat_c3, feat_c4, feat_c5, h_c2, w_c2, h_c3, w_c3, h_c4, w_c4, h_c5, w_c5,
        data_sampling_loc, data_attn_weight, batch_size, channels, num_views, num_query, num_point, data_col
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error in ms_deformable_im2col_cuda_c2345: %s\n", hipGetErrorString(err));
    }
}

void ms_deformable_im2col_cuda_c23456(
    const float* feat_c2,
    const float* feat_c3,
    const float* feat_c4,
    const float* feat_c5,
    const float* feat_c6,
    const int h_c2, const int w_c2,
    const int h_c3, const int w_c3,
    const int h_c4, const int w_c4,
    const int h_c5, const int w_c5,
    const int h_c6, const int w_c6,
    const float* data_sampling_loc,
    const float* data_attn_weight,
    const int batch_size,
    const int channels,
    const int num_views,
    const int num_query,
    const int num_point,
    float* data_col) {

    const int num_kernels = batch_size * num_query * channels;
    const int num_threads = CUDA_NUM_THREADS;

    ms_deformable_im2col_gpu_kernel_c23456 <<<GET_BLOCKS(num_kernels, num_threads), num_threads>>> (
        feat_c2, feat_c3, feat_c4, feat_c5, feat_c6, h_c2, w_c2, h_c3, w_c3, h_c4, w_c4, h_c5, w_c5, h_c6, w_c6,
        data_sampling_loc, data_attn_weight, batch_size, channels, num_views, num_query, num_point, data_col
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error in ms_deformable_im2col_cuda_c23456: %s\n", hipGetErrorString(err));
    }
}
